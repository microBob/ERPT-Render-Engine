#include "hip/hip_runtime.h"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "bugprone-reserved-identifier"
//
// Created by microbobu on 2/21/21.
//
#include "../include/optixLaunchParameters.h"
#include "optix_device.h"

/// Launch Parameters
extern "C" __constant__ OptixLaunchParameters optixLaunchParameters;

enum {
	SURFACE_RAY_TYPE = 0,
	RAY_TYPE_COUNT
};

/// Utility functions
__device__ float3 normalizeVectorGPU(float3 vector) {
	auto r_normal = rnorm3df(vector.x, vector.y, vector.z);

	return make_float3(vector.x * r_normal, vector.y * r_normal, vector.z * r_normal);
}

__device__ float3 vectorCrossProductGPU(float3 vectorA, float3 vectorB) {
	return make_float3(vectorA.y * vectorB.z - vectorA.z * vectorB.y, vectorA.z * vectorB.x - vectorA.x * vectorB.z,
	                   vectorA.x * vectorB.y - vectorA.y * vectorB.x);
}

__device__ float vectorDotProductGPU(float3 vectorA, float3 vectorB) {
	return vectorA.x * vectorB.x + vectorA.y * vectorB.y + vectorA.z * vectorB.z;
}

/// Payload management
static __forceinline__ __device__ void *unpackPointer(uint32_t i0, uint32_t i1) {
	const uint64_t rawPointer = static_cast<uint64_t>(i0) << 32 | i1;
	void *pointer = reinterpret_cast<void *>(rawPointer);
	return pointer;
}

static __forceinline__ __device__ void packPointer(void *pointer, uint32_t &i0, uint32_t &i1) {
	const auto rawPointer = reinterpret_cast<uint64_t>(pointer);
	i0 = rawPointer >> 32;
	i1 = rawPointer & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T *getPerRayData() {
	const uint32_t u0 = optixGetPayload_0();
	const uint32_t u1 = optixGetPayload_1();
	return reinterpret_cast<T *>( unpackPointer(u0, u1));
}

/// Ray generation program
extern "C" __global__ void __raygen__renderFrame() {
	// Get index and camera
	const unsigned int ix = optixGetLaunchIndex().x;
	const unsigned int iy = optixGetLaunchIndex().y;
	const auto &camera = optixLaunchParameters.camera;

	// Ray information storage
	float3 rayOrigin; // Where the ray starts
	float3 rayDirectionNormalized; // Where the ray goes

	if (optixLaunchParameters.systemState[MutationIndex] <
	    optixLaunchParameters.mutation.numberOfThem) { // Still have mutations = rendering
		/// Generate random ray direction
		// Get random number set
		const float3 &mutationNumbersSet = optixLaunchParameters.mutation.numbers[optixLaunchParameters.systemState[MutationIndex]];
		if (optixLaunchParameters.systemState[StartFromCameraBool]) {
			const unsigned int randScreenX = llrintf(
				mutationNumbersSet.x * static_cast<float>(optixLaunchParameters.frame.frameBufferSize.x - 1));
			const unsigned int randScreenY = llrintf(
				mutationNumbersSet.y * static_cast<float>(optixLaunchParameters.frame.frameBufferSize.y - 1));
			const auto screen = make_float2(
				(static_cast<float>(randScreenX) + 0.5f) /
				static_cast<float>(optixLaunchParameters.frame.frameBufferSize.x),
				(static_cast<float>(randScreenY) + 0.5f) /
				static_cast<float>(optixLaunchParameters.frame.frameBufferSize.y));
			auto screenMinus = make_float2(screen.x - 0.5f, screen.y - 0.5f);
			auto horizontalTimesScreenMinus = make_float3(screenMinus.x * camera.horizontal.x,
			                                              screenMinus.x * camera.horizontal.y,
			                                              screenMinus.x * camera.horizontal.z);
			auto verticalTimesScreenMinus = make_float3(screenMinus.y * camera.vertical.x,
			                                            screenMinus.y * camera.vertical.y,
			                                            screenMinus.y * camera.vertical.z);
			auto rawRayDirection = make_float3(
				camera.direction.x + horizontalTimesScreenMinus.x + verticalTimesScreenMinus.x,
				camera.direction.y + horizontalTimesScreenMinus.y + verticalTimesScreenMinus.y,
				camera.direction.z + horizontalTimesScreenMinus.z + verticalTimesScreenMinus.z);

			rayOrigin = camera.position;
			rayDirectionNormalized = normalizeVectorGPU(rawRayDirection);
		} else {
			RayHitMeta sourceRayMeta = optixLaunchParameters.rayHitMetas[optixLaunchParameters.systemState[RayHitMetaIndex]];

			const float3 newRayDirRaw = make_float3(sourceRayMeta.hitNormal.x - cospif(mutationNumbersSet.x),
			                                        sourceRayMeta.hitNormal.y - cospif(mutationNumbersSet.y),
			                                        sourceRayMeta.hitNormal.z - cospif(mutationNumbersSet.z / 2));
			const float rayDirInverseMagnitude = rnorm3df(newRayDirRaw.x, newRayDirRaw.y, newRayDirRaw.z);

			rayOrigin = sourceRayMeta.hitLocation;
			rayDirectionNormalized = make_float3(newRayDirRaw.x * rayDirInverseMagnitude,
			                                     newRayDirRaw.y * rayDirInverseMagnitude,
			                                     newRayDirRaw.z * rayDirInverseMagnitude);
		}

//		if (optixLaunchParameters.systemState[MutationIndex] == 2) {
//			printf("First Direction: <%f, %f, %f>\n", rayDirectionNormalized.x, rayDirectionNormalized.y,
//			       rayDirectionNormalized.z);
//		}

		optixLaunchParameters.systemState[MutationIndex]++;

		// Optix Trace
		optixTrace(optixLaunchParameters.optixTraversableHandle,
		           rayOrigin,
		           rayDirectionNormalized,
		           0.f,
		           1e20f,
		           0.0f,
		           OptixVisibilityMask(255),
		           OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		           SURFACE_RAY_TYPE,
		           RAY_TYPE_COUNT,
		           SURFACE_RAY_TYPE);
	} else { // Done rendering and is now checking for visibility
		// Create per ray data pointer
		colorVector pixelColorPerRayData;
		uint32_t payload0, payload1;
		packPointer(&pixelColorPerRayData, payload0, payload1);

		// Creating screen ray
		// TODO: use ix , iy as index of random numbers to pull from
		//
		const auto screen = make_float2(
			(static_cast<float>(ix) + 0.5f) /
			static_cast<float>(optixLaunchParameters.frame.frameBufferSize.x),
			(static_cast<float>(iy) + 0.5f) /
			static_cast<float>(optixLaunchParameters.frame.frameBufferSize.y));
		auto screenMinus = make_float2(screen.x - 0.5f, screen.y - 0.5f);
		auto horizontalTimesScreenMinus = make_float3(screenMinus.x * camera.horizontal.x,
		                                              screenMinus.x * camera.horizontal.y,
		                                              screenMinus.x * camera.horizontal.z);
		auto verticalTimesScreenMinus = make_float3(screenMinus.y * camera.vertical.x,
		                                            screenMinus.y * camera.vertical.y,
		                                            screenMinus.y * camera.vertical.z);
		auto rawRayDirection = make_float3(
			camera.direction.x + horizontalTimesScreenMinus.x + verticalTimesScreenMinus.x,
			camera.direction.y + horizontalTimesScreenMinus.y + verticalTimesScreenMinus.y,
			camera.direction.z + horizontalTimesScreenMinus.z + verticalTimesScreenMinus.z);

		rayOrigin = camera.position;
		rayDirectionNormalized = normalizeVectorGPU(rawRayDirection);

		// Do trace
		optixTrace(optixLaunchParameters.optixTraversableHandle,
		           rayOrigin,
		           rayDirectionNormalized,
		           0.f, // TODO: this is tmax
		           1e20f,
		           0.0f,
		           OptixVisibilityMask(255),
		           OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		           SURFACE_RAY_TYPE,
		           RAY_TYPE_COUNT,
		           SURFACE_RAY_TYPE,
		           payload0,
		           payload1);

		// Loop through recorded hits
		const unsigned int frameIndex = ix + iy * optixLaunchParameters.frame.frameBufferSize.x;
		float3 visibilityHitLocation = optixLaunchParameters.frame.visibleLocations[frameIndex];
		float energy = 0;

		if (visibilityHitLocation.x != nanf("")) {
			for (unsigned int hitIndex = 0;
			     hitIndex <= optixLaunchParameters.systemState[RayHitMetaIndex]; ++hitIndex) {
				RayHitMeta thisHitMeta = optixLaunchParameters.rayHitMetas[hitIndex];
				float3 rayHitLocation = thisHitMeta.hitLocation;
				float visibilityTolerance = 1 / static_cast<float>(optixLaunchParameters.systemState[VisibilityTolerance]);
				bool inXRange = fdimf(visibilityHitLocation.x, rayHitLocation.x) < visibilityTolerance;
				bool inYRange = fdimf(visibilityHitLocation.y, rayHitLocation.y) < visibilityTolerance;
				bool inZRange = fdimf(visibilityHitLocation.z, rayHitLocation.z) < visibilityTolerance;

				if (inXRange && inYRange && inZRange) {
					energy = thisHitMeta.energy;
					break;
				}
			}
		}

		// Edit pixelColorPerRayData and record
		pixelColorPerRayData = {pixelColorPerRayData.r * energy, pixelColorPerRayData.g * energy,
		                        pixelColorPerRayData.b * energy};
		optixLaunchParameters.frame.frameColorBuffer[frameIndex] = pixelColorPerRayData;
	}
}

/// Miss program
extern "C" __global__ void __miss__radiance() {
	if (optixLaunchParameters.systemState[MutationIndex] ==
	    optixLaunchParameters.mutation.numberOfThem) { // Visibility check operation
		const unsigned int ix = optixGetLaunchIndex().x;
		const unsigned int iy = optixGetLaunchIndex().y;
		const unsigned int visibleIndex = ix + iy * optixLaunchParameters.frame.frameBufferSize.x;
		optixLaunchParameters.frame.visibleLocations[visibleIndex] = make_float3(nanf(""), nanf(""), nanf(""));
	} else {
		// TODO: add to miss try counter, reset to camera if over limit
	}
}

/// Hit program
extern "C" __global__ void __closesthit__radiance() {
	const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
	const unsigned int ix = optixGetLaunchIndex().x;
	const unsigned int iy = optixGetLaunchIndex().y;

	// Essential hit data
	const float3 rayDir = optixGetWorldRayDirection();
	const float3 rayOrigin = optixGetWorldRayOrigin();
	const float rayLength = optixGetRayTmax();

	// Check if valid hit
	if (rayLength <= 0.0001) { // Basically 0
		printf("No ray length, skipping\n");
		return;
	}

	const float3 hitLocation = make_float3(rayOrigin.x + rayLength * rayDir.x, rayOrigin.y + rayLength * rayDir.y,
	                                       rayOrigin.z + rayLength * rayDir.z);

	// Surface normal
	const int primitiveIndex = optixGetPrimitiveIndex();
	const int3 index = sbtData.index[primitiveIndex];
	const float3 &vertexA = sbtData.vertex[index.x];
	const float3 &vertexB = sbtData.vertex[index.y];
	const float3 &vertexC = sbtData.vertex[index.z];
	auto vertexBMinusA = make_float3(vertexB.x - vertexA.x, vertexB.y - vertexA.y, vertexB.z - vertexA.z);
	auto vertexCMinusA = make_float3(vertexC.x - vertexA.x, vertexC.y - vertexA.y, vertexC.z - vertexA.z);
	const float3 surfaceNormal = normalizeVectorGPU(vectorCrossProductGPU(vertexBMinusA, vertexCMinusA));

	// Ray meta encode
	RayHitMeta thisRayHitMeta = {hitLocation, rayOrigin, surfaceNormal, rayLength, 1,
	                             optixLaunchParameters.systemState[StartFromCameraBool] == 1,
	                             optixLaunchParameters.systemState[RayHitMetaIndex]};


//	if (rayLength < 1) {
//		printf("Less than 1 rayLength: %f, %lu, %lu, (%f, %f, %f)\n", rayLength,
//		       optixLaunchParameters.systemState[MutationIndex],
//		       optixLaunchParameters.systemState[RayHitMetaIndex], hitLocation.x, hitLocation.y, hitLocation.z);
//	}

	if (optixLaunchParameters.systemState[MutationIndex] <
	    optixLaunchParameters.mutation.numberOfThem) { // Trace operation
		if (sbtData.kind == Mesh) {
			if (!optixLaunchParameters.systemState[RayHitMetaIndex] &&
			    optixLaunchParameters.systemState[StartFromCameraBool]) { // If == 0 and start from camera
//				printf(
//					"Index: %lu, Camera: %lu, Mutation: %lu | Ray Origin: (%f, %f, %f) | Hit Location: (%f, %f, %f) | Hit Normal: (%f, %f, %f)\n",
//					optixLaunchParameters.systemState[RayHitMetaIndex],
//					optixLaunchParameters.systemState[StartFromCameraBool],
//					optixLaunchParameters.systemState[MutationIndex], rayOrigin.x, rayOrigin.y, rayOrigin.z,
//					hitLocation.x, hitLocation.y, hitLocation.z, surfaceNormal.x, surfaceNormal.y, surfaceNormal.z);

				optixLaunchParameters.rayHitMetas[0] = thisRayHitMeta;
			} else {
//				if (optixLaunchParameters.systemState[RayHitMetaIndex] == 0) {
//					printf(
//						"Index: %lu, Camera: %lu, Mutation: %lu | Hit Location: (%f, %f, %f) | ray direction: (%f, %f, %f)\n",
//						optixLaunchParameters.systemState[RayHitMetaIndex],
//						optixLaunchParameters.systemState[StartFromCameraBool],
//						optixLaunchParameters.systemState[MutationIndex], hitLocation.x, hitLocation.y,
//						hitLocation.z, rayDir.x, rayDir.y, rayDir.z);
//				}
				optixLaunchParameters.systemState[RayHitMetaIndex]++;
				optixLaunchParameters.rayHitMetas[optixLaunchParameters.systemState[RayHitMetaIndex]] = thisRayHitMeta;
//				if (optixLaunchParameters.systemState[RayHitMetaIndex] == 1) {
//					printf("From: (%f, %f, %f), Source Index: %lu | hit length: %f\n", thisRayHitMeta.from.x,
//					       thisRayHitMeta.from.y,
//					       thisRayHitMeta.from.z, thisRayHitMeta.sourceRayIndex, rayLength);
//				}
			}

			if (optixLaunchParameters.systemState[StartFromCameraBool]) {
				optixLaunchParameters.systemState[StartFromCameraBool] = 0;
			}
		} else { // Hit a light source
//			printf("Hit Light at ray#%lu\n", optixLaunchParameters.systemState[RayHitMetaIndex]);
			// Directly apply if root ray
			if (optixLaunchParameters.systemState[StartFromCameraBool]) {
				thisRayHitMeta.energy = sbtData.energy / (rayLength * rayLength); // 1 / r^2

				if (!optixLaunchParameters.systemState[RayHitMetaIndex] &&
				    optixLaunchParameters.systemState[StartFromCameraBool]) { // If == 0 and start from camera
					optixLaunchParameters.rayHitMetas[0] = thisRayHitMeta;
				} else {
					optixLaunchParameters.systemState[RayHitMetaIndex]++;
					optixLaunchParameters.rayHitMetas[optixLaunchParameters.systemState[RayHitMetaIndex]] = thisRayHitMeta;
				}
			} else {
				// Reset next ray back to camera
				optixLaunchParameters.systemState[StartFromCameraBool] = 1;
				/// Cycle through each ray in this path
				unsigned long metaSearchIndex = optixLaunchParameters.systemState[RayHitMetaIndex];
				float lastEnergy = sbtData.energy; // Set energy to distribute
				// Loop through source rays until hit root
				while (!optixLaunchParameters.rayHitMetas[metaSearchIndex].isRootRay) {
					// Calculate 1 / r^2 from energy
					float searchedMetaRayLength = optixLaunchParameters.rayHitMetas[metaSearchIndex].rayLength;
					lastEnergy /= (searchedMetaRayLength * searchedMetaRayLength);
					// Set as energy
					optixLaunchParameters.rayHitMetas[metaSearchIndex].energy = lastEnergy;
					// Set next search index
					metaSearchIndex = optixLaunchParameters.rayHitMetas[metaSearchIndex].sourceRayIndex;
				}
			}
		}

	} else { // Visibility check operation
		const unsigned int visibleIndex = ix + iy * optixLaunchParameters.frame.frameBufferSize.x;
		optixLaunchParameters.frame.visibleLocations[visibleIndex] = hitLocation;

		colorVector &perRayData = *(colorVector *) getPerRayData<colorVector>();
		perRayData = {sbtData.color.r, sbtData.color.g, sbtData.color.b};
	}
}
extern "C" __global__ void __anyhit__radiance() {}

#pragma clang diagnostic pop