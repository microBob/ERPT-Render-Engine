#include "hip/hip_runtime.h"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "bugprone-reserved-identifier"
//
// Created by microbobu on 2/21/21.
//
#include "../include/optixLaunchParameters.h"
#include "optix_device.h"

/// Launch Parameters
extern "C" __constant__ OptixLaunchParameters optixLaunchParameters;

enum {
	SURFACE_RAY_TYPE = 0,
	RAY_TYPE_COUNT
};

/// Utility functions
__device__ float3 normalizeVectorGPU(float3 vector) {
	const auto r_normal = rnorm3df(vector.x, vector.y, vector.z);

	return make_float3(vector.x * r_normal, vector.y * r_normal, vector.z * r_normal);
}

__device__ float3 vectorCrossProductGPU(float3 vectorA, float3 vectorB) {
	return make_float3(vectorA.y * vectorB.z - vectorA.z * vectorB.y, vectorA.z * vectorB.x - vectorA.x * vectorB.z,
	                   vectorA.x * vectorB.y - vectorA.y * vectorB.x);
}

__device__ float vectorDotProductGPU(float3 vectorA, float3 vectorB) {
	return vectorA.x * vectorB.x + vectorA.y * vectorB.y + vectorA.z * vectorB.z;
}

/// Payload management
static __forceinline__ __device__ void *unpackPointer(uint32_t i0, uint32_t i1) {
	const uint64_t rawPointer = static_cast<uint64_t>(i0) << 32 | i1;
	void *pointer = reinterpret_cast<void *>(rawPointer);
	return pointer;
}

static __forceinline__ __device__ void packPointer(void *pointer, uint32_t &i0, uint32_t &i1) {
	const auto rawPointer = reinterpret_cast<uint64_t>(pointer);
	i0 = rawPointer >> 32;
	i1 = rawPointer & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T *getPerRayData() {
	const uint32_t u0 = optixGetPayload_0();
	const uint32_t u1 = optixGetPayload_1();
	return reinterpret_cast<T *>( unpackPointer(u0, u1));
}

/// Ray generation program
extern "C" __global__ void __raygen__renderFrame() {
	// Get index and camera
	const unsigned int ix = optixGetLaunchIndex().x;
	const unsigned int iy = optixGetLaunchIndex().y;
	const unsigned int mutationNumberIndex = ix + iy * optixLaunchParameters.frame.frameBufferSize.x;
	unsigned int screenX = llrintf(static_cast<float>(optixLaunchParameters.frame.frameBufferSize.x) *
	                               optixLaunchParameters.curMutationNumbers[mutationNumberIndex]);
	unsigned int screenY = llrintf(static_cast<float>(optixLaunchParameters.frame.frameBufferSize.y) *
	                               optixLaunchParameters.curMutationNumbers[mutationNumberIndex + 1]);
	unsigned int pixelIndex = screenX + screenY * optixLaunchParameters.frame.frameBufferSize.x;

	const auto &camera = optixLaunchParameters.camera;

	/// Starting ray from camera
	// Create per ray data
	PerRayData rayData;
	uint32_t payload0, payload1;
	packPointer(&rayData, payload0, payload1);

	// Create base screen ray
	const auto screen = make_float2(
		(static_cast<float>(screenX) + 0.5f) /
		static_cast<float>(optixLaunchParameters.frame.frameBufferSize.x),
		(static_cast<float>(screenY) + 0.5f) /
		static_cast<float>(optixLaunchParameters.frame.frameBufferSize.y));
	auto screenMinus = make_float2(screen.x - 0.5f, screen.y - 0.5f);
	auto horizontalTimesScreenMinus = make_float3(screenMinus.x * camera.horizontal.x,
	                                              screenMinus.x * camera.horizontal.y,
	                                              screenMinus.x * camera.horizontal.z);
	auto verticalTimesScreenMinus = make_float3(screenMinus.y * camera.vertical.x,
	                                            screenMinus.y * camera.vertical.y,
	                                            screenMinus.y * camera.vertical.z);
	auto rawRayDirection = make_float3(
		camera.direction.x + horizontalTimesScreenMinus.x + verticalTimesScreenMinus.x,
		camera.direction.y + horizontalTimesScreenMinus.y + verticalTimesScreenMinus.y,
		camera.direction.z + horizontalTimesScreenMinus.z + verticalTimesScreenMinus.z);

	float3 rayOrigin = camera.position;
	float3 rayDirectionNormalized = normalizeVectorGPU(rawRayDirection);
	atomicAdd(&optixLaunchParameters.pixelVisits[pixelIndex], 1);

	// Trace
	optixTrace(optixLaunchParameters.optixTraversableHandle,
	           rayOrigin,
	           rayDirectionNormalized,
	           0.001f, // Needs to have gone somewhere
	           1e20f,
	           0.0f,
	           OptixVisibilityMask(255),
	           OPTIX_RAY_FLAG_DISABLE_ANYHIT,
	           SURFACE_RAY_TYPE,
	           RAY_TYPE_COUNT,
	           SURFACE_RAY_TYPE,
	           payload0,
	           payload1);

	colorVector baseColor;
	bool raySuccessful;
	if (rayData.normal.x + rayData.normal.y + rayData.normal.z != 0) {
		baseColor = rayData.color;

		// Increment Energy at pixel if a light source was hit
		if (rayData.light) {
			raySuccessful = true;
		} else { // Else, continue with second ray
			/// Reflected ray
			for (int depthIndex = 0; depthIndex < optixLaunchParameters.traceDepth; ++depthIndex) {
				// Create ray
				const float r = sqrt(
					optixLaunchParameters.curMutationNumbers[mutationNumberIndex + 2 + depthIndex * 2]);
				const float phi = 2 * 3.1415f *
				                  optixLaunchParameters.curMutationNumbers[mutationNumberIndex + 3 + depthIndex * 2];
				const float circleX = r * cos(phi);
				const float circleY = r * sin(phi);
				const float circleZ = sqrt(1 - (r * r));
				const float3 newDirection = make_float3(
					rayData.xAxis.x * circleX + rayData.yAxis.x * circleY + rayData.normal.x * circleZ,
					rayData.xAxis.y * circleX + rayData.yAxis.y * circleY + rayData.normal.y * circleZ,
					rayData.xAxis.z * circleX + rayData.yAxis.z * circleY + rayData.normal.z * circleZ);

				rayOrigin = rayData.location;
				rayDirectionNormalized = newDirection;//normalizeVectorGPU(newDirection);


				// Trace
				optixTrace(optixLaunchParameters.optixTraversableHandle,
				           rayOrigin,
				           rayDirectionNormalized,
				           0.001f, // Needs to have gone somewhere
				           1e20f,
				           0.0f,
				           OptixVisibilityMask(255),
				           OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				           SURFACE_RAY_TYPE,
				           RAY_TYPE_COUNT,
				           SURFACE_RAY_TYPE,
				           payload0,
				           payload1);

				// Stop if there was a miss
				if (rayData.normal.x + rayData.normal.y + rayData.normal.z == 0) {
					break;
				}
				// If there's light, increment data
				if (rayData.light) {
					raySuccessful = true;
					atomicAdd(&optixLaunchParameters.energyPerPixel[pixelIndex], rayData.energy);
					break;
				}
			}
		}

		if (raySuccessful) {
			const float colorSum =
				(baseColor.r + baseColor.g + baseColor.b) / rayData.energy *
				static_cast<float>(optixLaunchParameters.samples.total);
			atomicAdd(&optixLaunchParameters.frame.frameColorBuffer[pixelIndex].r, baseColor.r / colorSum);
			atomicAdd(&optixLaunchParameters.frame.frameColorBuffer[pixelIndex].g, baseColor.g / colorSum);
			atomicAdd(&optixLaunchParameters.frame.frameColorBuffer[pixelIndex].b, baseColor.b / colorSum);
		}
	}
}

/// Miss program
extern "C" __global__ void __miss__radiance() {
	PerRayData &perRayData = *(PerRayData *) getPerRayData<PerRayData>();
	const auto zeroVector = make_float3(0, 0, 0);
	perRayData = {zeroVector, zeroVector, zeroVector, zeroVector, {}, 0, false};
}

/// Hit program
extern "C" __global__ void __closesthit__radiance() {
	const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();

	// Essential hit data
	const float3 rayDir = optixGetWorldRayDirection();
	const float3 rayOrigin = optixGetWorldRayOrigin();
	const float rayLength = optixGetRayTmax();

	// Check if valid hit
	if (rayLength <= 0.0001) { // Basically 0
		printf("No ray length, skipping\n");
		return;
	}

	const float3 hitLocation = make_float3(rayOrigin.x + rayLength * rayDir.x, rayOrigin.y + rayLength * rayDir.y,
	                                       rayOrigin.z + rayLength * rayDir.z);

	// Surface normal
	const int primitiveIndex = optixGetPrimitiveIndex();
	const int3 index = sbtData.index[primitiveIndex];
	const float3 &vertexA = sbtData.vertex[index.x];
	const float3 &vertexB = sbtData.vertex[index.y];
	const float3 &vertexC = sbtData.vertex[index.z];
	const auto vertexBMinusA = make_float3(vertexB.x - vertexA.x, vertexB.y - vertexA.y, vertexB.z - vertexA.z);
	const auto vertexCMinusA = make_float3(vertexC.x - vertexA.x, vertexC.y - vertexA.y, vertexC.z - vertexA.z);
	const float3 normalAxis = normalizeVectorGPU(vectorCrossProductGPU(vertexBMinusA, vertexCMinusA));
	const colorVector normalColor = {(normalAxis.x + 1) / 2, (normalAxis.y + 1) / 2, (normalAxis.z + 1) / 2};

	// Second Axis
	// TODO: causing bad indirect lighting
	const unsigned int ix = optixGetLaunchIndex().x;
	const unsigned int iy = optixGetLaunchIndex().y;
	const unsigned int mutationNumberIndex = ix + iy * optixLaunchParameters.frame.frameBufferSize.x;
	const float3 yAxis = normalizeVectorGPU(
		vectorCrossProductGPU(make_float3(optixLaunchParameters.curMutationNumbers[mutationNumberIndex],
		                                  optixLaunchParameters.curMutationNumbers[mutationNumberIndex + 1],
		                                  optixLaunchParameters.curMutationNumbers[mutationNumberIndex + 2]),
		                      normalAxis));
//	const float3 yAxis = normalizeVectorGPU(vectorCrossProductGPU(optixLaunchParameters.camera.direction, normalAxis));

	// Third Axis
	const float3 xAxis = normalizeVectorGPU(vectorCrossProductGPU(normalAxis, yAxis));

	// Encode per ray data
	PerRayData &perRayData = *(PerRayData *) getPerRayData<PerRayData>();
	perRayData = {hitLocation, normalAxis, xAxis, yAxis, sbtData.color, sbtData.energy, sbtData.kind == Light};
}
extern "C" __global__ void __anyhit__radiance() {}

#pragma clang diagnostic pop