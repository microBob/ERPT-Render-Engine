#include "hip/hip_runtime.h"
//
// Created by microbobu on 1/10/21.
//
#include "../include/transformations.cuh"


float *Transformations::get_worldToCameraMatrix() {
	return worldToCameraMatrix;
}

void Transformations::set_worldToCameraMatrix(float x, float y, float z, float degX, float degY, float degZ) {
	// Convert degrees to radians and negate
	float radX = -degX * (float) M_PI / 180.0f;
	float radY = -degY * (float) M_PI / 180.0f;
	float radZ = -degZ * (float) M_PI / 180.0f;

	/// Copy in matrix
	// 1
	worldToCameraMatrix[0] = cos(radY) * cos(radZ);
	worldToCameraMatrix[1] = sin(radX) * sin(radY) * cos(radZ) + cos(radX) * sin(radZ);
	worldToCameraMatrix[2] = sin(radX) * sin(radZ) - cos(radX) * sin(radY) * cos(radZ);
	// 2
	worldToCameraMatrix[4] = -cos(radY) * sin(radZ);
	worldToCameraMatrix[5] = cos(radX) * cos(radZ) - sin(radX) * sin(radY) * sin(radZ);
	worldToCameraMatrix[6] = cos(radX) * sin(radY) * sin(radZ) + sin(radX) * cos(radZ);
	// 3
	worldToCameraMatrix[8] = sin(radY);
	worldToCameraMatrix[9] = -sin(radX) * cos(radY);
	worldToCameraMatrix[10] = cos(radX) * cos(radY);
	// 4
	worldToCameraMatrix[12] = y * cos(radY) * sin(radZ) - x * cos(radY) * cos(radZ) - z * sin(radY);
	worldToCameraMatrix[13] =
		z * sin(radX) * cos(radY) - x * (sin(radX) * sin(radY) * cos(radZ) + cos(radX) * sin(radZ)) -
		y * (cos(radX) * cos(radZ) - sin(radX) * sin(radY) * sin(radZ));
	worldToCameraMatrix[14] = -x * (sin(radX) * sin(radZ) - cos(radX) * sin(radY) * cos(radZ)) -
	                          y * (cos(radX) * sin(radY) * sin(radZ) + sin(radX) * cos(radZ)) -
	                          z * cos(radX) * cos(radY);
	worldToCameraMatrix[15] = 1.0f;
}

void Transformations::set_perspectiveMatrix(float screenWidth, float screenHeight, float fovRadians, float zFar,
                                            float zNear) {
	perspectiveMatrix[0] = screenWidth / screenHeight / tan(fovRadians / 2);
	perspectiveMatrix[5] = 1.0f / tan(fovRadians / 2);
	perspectiveMatrix[10] = 2.0f / (zFar - zNear);
	perspectiveMatrix[11] = -1.0f;
	perspectiveMatrix[14] = -(zNear - zFar) / (zFar + zNear);
}

float *Transformations::get_perspectiveMatrix() {
	return perspectiveMatrix;
}

void Transformations::convertVerticesToCameraSpace(float *vertices, const int vertexCount) {
	/// Expand worldToCameraMatrix
	// Define and malloc expanded matrix
	float *expandedWorldToCameraMatrix;
	expandedMatrixByteSize = vertexCount * matrixByteSize;
	hipMallocManaged(&expandedWorldToCameraMatrix, expandedMatrixByteSize);
	hipMemPrefetchAsync(expandedWorldToCameraMatrix, expandedMatrixByteSize, k.get_cpuID());
	// Copy
	for (int i = 0; i < vertexCount; i += 16) {
		copy(worldToCameraMatrix, worldToCameraMatrix + 16, expandedWorldToCameraMatrix + i);
	}
	// Switch to GPU
	hipMemAdvise(expandedWorldToCameraMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetPreferredLocation,
	              k.get_gpuID());
	hipMemAdvise(expandedWorldToCameraMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetReadMostly,
	              k.get_gpuID());
	hipMemPrefetchAsync(expandedWorldToCameraMatrix, vertexCount * expandedMatrixByteSize, k.get_gpuID());

	/// Initialize cameraVertices
	hipMallocManaged(&cameraVertices, vertexCount * sizeof(float));
	hipMemAdvise(cameraVertices, vertexCount * sizeof(float), hipMemAdviseSetPreferredLocation, k.get_gpuID());
	hipMemPrefetchAsync(cameraVertices, vertexCount * sizeof(float), k.get_gpuID());

	/// cuBLAS
	status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 1, 4, &alpha, expandedWorldToCameraMatrix,
	                                   4, 16, vertices, 4, 4, &beta, cameraVertices, 4, 4, vertexCount);
	hipDeviceSynchronize();
	assert(status == HIPBLAS_STATUS_SUCCESS);

	// Cleanup
	hipFree(expandedWorldToCameraMatrix);
}

void Transformations::convertToScreenSpace(const int vertexCount) {
	/// Expand perspectiveMatrix
	// Define and malloc expanded matrix
	float *expandedPerspectiveMatrix;
	hipMallocManaged(&expandedPerspectiveMatrix, expandedMatrixByteSize);
	hipMemPrefetchAsync(expandedPerspectiveMatrix, expandedMatrixByteSize, k.get_cpuID());
	// Copy
	for (int i = 0; i < vertexCount; i += 16) {
		copy(perspectiveMatrix, perspectiveMatrix + 16, expandedPerspectiveMatrix + i);
	}
	// Switch to GPU
	hipMemAdvise(expandedPerspectiveMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetPreferredLocation,
	              k.get_gpuID());
	hipMemAdvise(expandedPerspectiveMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetReadMostly,
	              k.get_gpuID());
	hipMemPrefetchAsync(expandedPerspectiveMatrix, vertexCount * expandedMatrixByteSize, k.get_gpuID());

	/// Initialize screenVertices
	hipMallocManaged(&screenVertices, vertexCount * sizeof(float));
	hipMemAdvise(screenVertices, vertexCount * sizeof(float), hipMemAdviseSetPreferredLocation, k.get_gpuID());
	hipMemPrefetchAsync(screenVertices, vertexCount * sizeof(float), k.get_gpuID());

	/// cuBLAS
	status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 1, 4, &alpha, expandedPerspectiveMatrix,
	                                   4, 16, cameraVertices, 4, 4, &beta, screenVertices, 4, 4, vertexCount);
	hipDeviceSynchronize();
	assert(status == HIPBLAS_STATUS_SUCCESS);

	// Cleanup
	hipFree(expandedPerspectiveMatrix);
}
