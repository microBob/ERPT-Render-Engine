#include "hip/hip_runtime.h"
//
// Created by microbobu on 1/10/21.
//
#include "../include/transformations.cuh"


float *Transformations::get_worldToCameraMatrix() {
	return worldToCameraMatrix;
}

void Transformations::set_worldToCameraMatrix(float x, float y, float z, float degX, float degY, float degZ) {
	// Convert degrees to radians and negate
	float radX = -degX * (float) M_PI / 180.0f;
	float radY = -degY * (float) M_PI / 180.0f;
	float radZ = -degZ * (float) M_PI / 180.0f;

	/// Copy in matrix
	// 1
	worldToCameraMatrix[0] = cos(radY) * cos(radZ);
	worldToCameraMatrix[1] = sin(radX) * sin(radY) * cos(radZ) + cos(radX) * sin(radZ);
	worldToCameraMatrix[2] = sin(radX) * sin(radZ) - cos(radX) * sin(radY) * cos(radZ);
	// 2
	worldToCameraMatrix[4] = -cos(radY) * sin(radZ);
	worldToCameraMatrix[5] = cos(radX) * cos(radZ) - sin(radX) * sin(radY) * sin(radZ);
	worldToCameraMatrix[6] = cos(radX) * sin(radY) * sin(radZ) + sin(radX) * cos(radZ);
	// 3
	worldToCameraMatrix[8] = sin(radY);
	worldToCameraMatrix[9] = -sin(radX) * cos(radY);
	worldToCameraMatrix[10] = cos(radX) * cos(radY);
	// 4
	worldToCameraMatrix[12] = y * cos(radY) * sin(radZ) - x * cos(radY) * cos(radZ) - z * sin(radY);
	worldToCameraMatrix[13] =
		z * sin(radX) * cos(radY) - x * (sin(radX) * sin(radY) * cos(radZ) + cos(radX) * sin(radZ)) -
		y * (cos(radX) * cos(radZ) - sin(radX) * sin(radY) * sin(radZ));
	worldToCameraMatrix[14] = -x * (sin(radX) * sin(radZ) - cos(radX) * sin(radY) * cos(radZ)) -
	                          y * (cos(radX) * sin(radY) * sin(radZ) + sin(radX) * cos(radZ)) -
	                          z * cos(radX) * cos(radY);
	worldToCameraMatrix[15] = 1.0f;
}

void Transformations::set_perspectiveMatrix(float screenWidth, float screenHeight, float fovRadians, float zFar,
                                            float zNear) {
	perspectiveMatrix[0] = screenWidth / screenHeight / tan(fovRadians / 2);
	perspectiveMatrix[5] = 1.0f / tan(fovRadians / 2);
	perspectiveMatrix[10] = 2.0f / (zFar - zNear);
	perspectiveMatrix[11] = -1.0f;
	perspectiveMatrix[14] = -(zNear - zFar) / (zFar + zNear);
}

float *Transformations::get_perspectiveMatrix() {
	return perspectiveMatrix;
}

void Transformations::convertVerticesToCameraSpace(float *vertices, const int vertexCount) {
	/// Expand worldToCameraMatrix
	// Define and malloc expanded matrix
	float *expandedWorldToCameraMatrix;
	expandedMatrixByteSize = vertexCount * matrixByteSize;
	hipMallocManaged(&expandedWorldToCameraMatrix, expandedMatrixByteSize);
	hipMemPrefetchAsync(expandedWorldToCameraMatrix, expandedMatrixByteSize, k.get_cpuID());
	// Copy
	for (int i = 0; i < vertexCount; i += 16) {
		copy(worldToCameraMatrix, worldToCameraMatrix + 16, expandedWorldToCameraMatrix + i);
	}
	// Switch to GPU
	hipMemAdvise(expandedWorldToCameraMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetPreferredLocation,
	              k.get_gpuID());
	hipMemAdvise(expandedWorldToCameraMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetReadMostly,
	              k.get_gpuID());
	hipMemPrefetchAsync(expandedWorldToCameraMatrix, vertexCount * expandedMatrixByteSize, k.get_gpuID());

	/// Initialize cameraVertices
	hipMallocManaged(&cameraVertices, vertexCount * sizeof(float));
	hipMemAdvise(cameraVertices, vertexCount * sizeof(float), hipMemAdviseSetPreferredLocation, k.get_gpuID());
	hipMemPrefetchAsync(cameraVertices, vertexCount * sizeof(float), k.get_gpuID());

	/// cuBLAS
	status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 1, 4, &alpha, expandedWorldToCameraMatrix,
	                                   4, 16, vertices, 4, 4, &beta, cameraVertices, 4, 4, vertexCount);
	hipDeviceSynchronize();
	assert(status == HIPBLAS_STATUS_SUCCESS);

	// Cleanup
	hipFree(expandedWorldToCameraMatrix);
}

void Transformations::convertToPerspectiveSpace(const int vertexCount) {
	/// Expand perspectiveMatrix
	// Define and malloc expanded matrix
	float *expandedPerspectiveMatrix;
	hipMallocManaged(&expandedPerspectiveMatrix, expandedMatrixByteSize);
	hipMemPrefetchAsync(expandedPerspectiveMatrix, expandedMatrixByteSize, k.get_cpuID());
	// Copy
	for (int i = 0; i < vertexCount; i += 16) {
		copy(perspectiveMatrix, perspectiveMatrix + 16, expandedPerspectiveMatrix + i);
	}
	// Switch to GPU
	hipMemAdvise(expandedPerspectiveMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetPreferredLocation,
	              k.get_gpuID());
	hipMemAdvise(expandedPerspectiveMatrix, vertexCount * expandedMatrixByteSize, hipMemAdviseSetReadMostly,
	              k.get_gpuID());
	hipMemPrefetchAsync(expandedPerspectiveMatrix, vertexCount * expandedMatrixByteSize, k.get_gpuID());

	/// Initialize perspectiveVertices
	hipMallocManaged(&perspectiveVertices, vertexCount * sizeof(float));
	hipMemAdvise(perspectiveVertices, vertexCount * sizeof(float), hipMemAdviseSetPreferredLocation, k.get_gpuID());
	hipMemPrefetchAsync(perspectiveVertices, vertexCount * sizeof(float), k.get_gpuID());

	/// cuBLAS
	status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 1, 4, &alpha, expandedPerspectiveMatrix,
	                                   4, 16, cameraVertices, 4, 4, &beta, perspectiveVertices, 4, 4, vertexCount);
	hipDeviceSynchronize();
	assert(status == HIPBLAS_STATUS_SUCCESS);

	// Cleanup
	hipFree(expandedPerspectiveMatrix);
}

__global__ void
convertToScreenSpace(float *input, const int vertexCount, float *output, float screenWidth, float screenHeight) {
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int increment = blockDim.x * gridDim.x;

	while (tid < vertexCount) {
		// Skip if point is outside of z-cull space
		if (abs(input[sceneToLinearGPU(tid, 2, 0)]) > 1) {
			tid += increment;
			continue;
		}
		// Skip if point will cause divide by 0
		if (input[sceneToLinearGPU(tid, 3, 0)] == 0) {
			tid += increment;
			continue;
		}

		// Calculate final x and y
		output[sceneToLinearGPU(tid, 0, 2)] =
			input[sceneToLinearGPU(tid, 0, 4)] / input[sceneToLinearGPU(tid, 3, 4)] * screenWidth / 2.0f;
		output[sceneToLinearGPU(tid, 1, 2)] =
			input[sceneToLinearGPU(tid, 1, 4)] / input[sceneToLinearGPU(tid, 3, 4)] * screenHeight / 2.0f;

		// Increment tid
		tid += increment;
	}
}

__device__ unsigned int sceneToLinearGPU(unsigned int vertex, int coordinate, int dim) {
	return vertex * dim + coordinate;
}
